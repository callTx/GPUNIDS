#include "hip/hip_runtime.h"
#include"WMGPUinspection.h"

void WMGPUInspection::exec(std::shared_ptr<Packet> pkt)
{



}


WMGPUInspection::WMGPUInspection()
{

}

WMGPUInspection::~WMGPUInspection()
{
}

void WMGPUInspection::cuda_sample_random_ints(int* x,int N){

	for(int i=0; i<N;i++){
		x[i]=rand();
	}
}

void WMGPUInspection::cuda_sample_print_ints(int* x,int N){

	for(int i=0; i<N;i++){
		std::cout<<"x["<<i<<"]: "<<x[i]<<std::endl;
	}
}

__global__ void add(int *a,int *b,int *c,int n){
	//*c= *a+ *b; //Heterogeneous computing
	//c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x]; //Blocks
	//c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x]; //Threads
	
	int index = threadIdx.x + blockIdx.x * blockDim.x; //
	if(index < n) c[index] = a[index] + b[index];
}

int WMGPUInspection::buildMatchingMachine(std::string rules, int bufferSize, int nThreads)
{
		std::cout<<"nThreads: "<<nThreads<<std::endl;
		
		// (14-15/05) getilne -- TmhMenorPadrao() -- padroes.size() -- TmhTodosPadroes() -- BlcStr -- B2=B --  tabeleIdxHash() -- vecPermu() -- TabelaHash() 

		//TODO (16/05) Copiar todo WMCPUInspection.cpp para WMGPUInspection.cu -- Paralelizar o "AdcTodosPadroesTabela()"

		//TODO (17/05) "for (){listPadroes[lp]}"

		//TODO (18/05)  ??? "listPotenPadr = tabelaPREFIX[j]"

		//TODO (19/05) "MathPadroes"

		//TODO (20/05) "AdcTodosPadroesTabela()"

		//TODO (21/05) "for(..){listPadroes[lp]}"

		//TODO (22/05) ??? "listPotenPadr = tabelaPREFIX[j]"

		//TODO (23/05) "MatchPadroes"


		/*
		int *a,*b,*c;
		//int a,b,c;
		int *d_a,*d_b,*d_c; 
		// size= sizeof(int);
		//std::cout<<"N: "<<N<<std::endl;
		int size = N * sizeof(int);

		//Aloca espaco para o device copiar de a,b,c
		hipMalloc( (void **)&d_a,size );		
		hipMalloc( (void **)&d_b,size );		
		hipMalloc( (void **)&d_c,size );
		
		a = (int*)malloc(size);
		cuda_sample_random_ints(a,N);
		b= (int*)malloc(size);
		cuda_sample_random_ints(b,N);
		c= (int*)malloc(size);


		//a =2;
		//b =7;
		//copia a,b para o device
		hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);		
		hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);	

		//Launch o kernel add() na GPU
		//add<<<N,1>>>(d_a,d_b,d_c); //blocks
		//add<<<1,N>>>(d_a,d_b,d_c); //threads

		
		//	@ N/THREADS_PER_BLOCK  = blocks number
		
		//add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a,d_b,d_c); //threads_per_block with vector multiple of blockDim.x = M 

		add<<<(N+M-1)/M, M >>>(d_a,d_b,d_c,N); //threads_per_block with vector which is not multiple of the blockDim.x = M	

		//Copiar os resultados de volta p/ host
		hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

		//cuda_sample_print_ints(c,N);
		//std::cout<<"c: "<<c<<std::endl;
		
		//Cleanup
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);

		//char *hipGetErrorString(hipError_t);
		//printf("%s\n",hipGetErrorString(hipGetLastError()));

		return 0;
		*/
}
